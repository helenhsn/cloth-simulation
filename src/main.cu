#include "glad.h"
#include "../include/mesh.hcu"
#include "../include/camera.h"
#include "../include/simulation.h"
#include <GLFW/glfw3.h>
#include <iostream>

using namespace std;
const unsigned int w_height = 1000;
const unsigned int w_width = 1000;
float lastX;
float lastY;
bool firstMouse = true;

// callbacks
void window_resize_callback(GLFWwindow *window, int width, int height);
void mouse_callback(GLFWwindow* window, double xpos, double ypos);
void keyboard_callback(GLFWwindow *window);


//Camera camera(glm::vec3(21.8819, 20.3187, 83.4559));
Camera camera(glm::vec3(70.8819, 20.3187, 900.0));

float dt = 0.0f;	// time between current frame and last frame
float lastFrame = 0.0f;
int main()
{
    // initializing OpenGL context using GLFW & GLAD
    if (!glfwInit())
    {
        std::cout << "Failed to init GLFW!" << std::endl;
        glfwTerminate();
        return -1;
    }
    GLFWwindow *window = glfwCreateWindow(w_width, w_height, "Cloth Simulation", 0, nullptr);
    if(!window)
    {
        std::cout << "Failed to create GLFW window!" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, window_resize_callback);
    glfwSetCursorPosCallback(window, mouse_callback);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }


    std::cout << "MAIN" << std::endl << std::flush;


    // setting up CUDA context

    int device_count;
    cudaErrorCheck(hipGetDeviceCount(&device_count));
    if (device_count == 0) 
    {
        std::cout << "No CUDA devices found!" << std::endl << std::flush ;
        return -1;
    }

    // setting up simulation

    int N = 128;
    ShaderProgram sphere_pgrm("shaders/cloth.vs", "shaders/cloth.fs");
    ShaderProgram cloth_pgrm("shaders/cloth.vs", "shaders/cloth.fs");
    Plane *grid = new Plane(cloth_pgrm, N);
    Sphere *sphere = new Sphere(sphere_pgrm, 10.0f, 50, 50);
    Simulation sim(grid);

    // Rendering options
    glEnable(GL_DEPTH_TEST);
    //glEnable(GL_CULL_FACE);
    //glCullFace(GL_BACK);

    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glClearColor(0.0, 0.0, 0.0, 1.0);

        float currentFrame = static_cast<float>(glfwGetTime());
        dt = currentFrame - lastFrame;
        lastFrame = currentFrame;

        keyboard_callback(window);

        // run simulation 
        sim.run(currentFrame);

        glm::mat4 projection = glm::perspective(glm::radians(camera.Zoom), (float)w_width / (float)w_height, 0.1f, 100000.0f);
        glm::mat4 view = camera.GetViewMatrix();
        //glm::mat4 model = glm::rotate(glm::mat4(1.0f), glm::radians(90.0f), glm::vec3(0.0f, 0.0f, 1.0f));
        glm::mat4 model = glm::mat4(1.0f);
        sphere_pgrm.use();
        sphere_pgrm.setMat4("projection", projection);
        sphere_pgrm.setMat4("view", view);
        //pgrm.setMat4("model", glm::rotate(model, 0.0f, glm::vec3(0.0, 1.0, 0.0)));
        sphere_pgrm.setMat4("model", glm::scale(model, glm::vec3(2.0f)));

        cloth_pgrm.use();
        cloth_pgrm.setMat4("projection", projection);
        cloth_pgrm.setMat4("view", view);
        //pgrm.setMat4("model", glm::rotate(model, 0.0f, glm::vec3(0.0, 1.0, 0.0)));
        cloth_pgrm.setMat4("model", glm::scale(model, glm::vec3(2.0f)));

        glPolygonMode(GL_FRONT_AND_BACK,  GL_LINE);
        grid->draw();
        glPolygonMode(GL_FRONT_AND_BACK,  GL_FILL);
        sphere->draw();

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glfwTerminate();
    return 0;

}

void window_resize_callback(GLFWwindow *window, int width, int height)
{
    glViewport(0, 0, width, height);
} 

// -------------------------------------------------------
void mouse_callback(GLFWwindow* window, double xposIn, double yposIn)
{
    float xpos = static_cast<float>(xposIn);
    float ypos = static_cast<float>(yposIn);

    if (firstMouse)
    {
        lastX = xpos;
        lastY = ypos;
        firstMouse = false;
        return;
    }

    float xoffset = xpos - lastX;
    float yoffset = lastY - ypos;

    lastX = xpos;
    lastY = ypos;

    camera.ProcessMouseMovement(xoffset, yoffset);
}

void keyboard_callback(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        camera.ProcessKeyboard(FORWARD, dt);
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        camera.ProcessKeyboard(BACKWARD, dt);
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        camera.ProcessKeyboard(LEFT, dt);
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        camera.ProcessKeyboard(RIGHT, dt);
     if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
        camera.ProcessKeyboard(UP, dt);
}