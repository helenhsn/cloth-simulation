#include "glad.h"
#include "../include/mesh.hcu"
#include "../include/camera.h"
#include "../include/simulation.h"
#include "../include/ui.h"
#include <iostream>

using namespace std;
float w_height = 1000.0;
float w_width = 1500.0;
float lastX;
float lastY;
bool firstMouse = true;

// callbacks
void windowResizeCallback(GLFWwindow *window, int width, int height);
void mouseCallback(GLFWwindow* window, double xpos, double ypos);
void keyboard_callback(GLFWwindow *window);


//Camera camera(glm::vec3(21.8819, 20.3187, 83.4559));
Camera camera(glm::vec3(2.0, 10., 20.0));

float dt = 0.0f;	// time between current frame and last frame
float lastFrame = 0.0f;
int main()
{
    // initializing OpenGL context using GLFW & GLAD
    if (!glfwInit())
    {
        std::cout << "Failed to init GLFW!" << std::endl;
        glfwTerminate();
        return -1;
    }
    GLFWwindow *window = glfwCreateWindow(static_cast<int>(w_width), static_cast<int>(w_height), "Cloth Simulation", 0, nullptr);
    if(!window)
    {
        std::cout << "Failed to create GLFW window!" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, windowResizeCallback);
    glfwSetCursorPosCallback(window, mouseCallback);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }


    std::cout << "MAIN" << std::endl << std::flush;


    // setting up CUDA context

    int device_count;
    cudaErrorCheck(hipGetDeviceCount(&device_count));
    if (device_count == 0) 
    {
        std::cout << "No CUDA devices found!" << std::endl << std::flush ;
        return -1;
    }


    // setting simulation objects (drawables)
    ShaderProgram ground_pgrm("../shaders/ground.vs", "../shaders/ground.fs");
    ShaderProgram simple_pgrm("../shaders/sphere.vs", "../shaders/sphere.fs");
    ShaderProgram cloth_pgrm("../shaders/cloth.vs", "../shaders/cloth.fs");


    glm::mat4x4 modelCloth = glm::scale(glm::mat4(1.0f), glm::vec3(0.08f, 1.0f, 0.08f));
    modelCloth = glm::translate(modelCloth, glm::vec3(0.0f, 6.0f, 0.0f));
    Plane *cloth = new Plane(cloth_pgrm.glid, modelCloth, 128);
    
    glm::mat4 scaleGround = glm::scale(glm::mat4(1.0f), 3000.0f*glm::vec3(1.0f, 0.0f, 1.0f));
    glm::mat4 modelGround = glm::translate(glm::mat4(1.0f), -1000.0f*glm::vec3(1.0f, 0.0f, 1.0f))*scaleGround;
    Plane *ground = new Plane(ground_pgrm.glid, modelGround, 50, {false, false, false});
    
    glm::mat4 modelSphere = glm::translate(glm::mat4(1.0f), glm::vec3(5.0f, 3.0f, 5.0f));
    Sphere *sphere = new Sphere(simple_pgrm.glid, modelSphere, 2.0);
    Simulation *sim = new Simulation(cloth);

    glm::mat4 modelSimpleCollider = glm::translate(glm::mat4(1.0f), glm::vec3(2.0f, 8.0f, 2.0f));
    Plane *simpleCollider = new Plane(simple_pgrm.glid, modelSimpleCollider, 10);

    glm::mat4 scaleCollider = glm::scale(glm::mat4(1.0f), 0.03f*glm::vec3(1.0f, 1.0f, 1.0f));
    glm::mat4 transCollider = glm::translate(glm::mat4(1.0f), glm::vec3(5.0f, 3.0f, 5.0f));
    glm::mat4 modelCollider = transCollider*scaleCollider;
    MeshFromPLY *anotherCollider = new MeshFromPLY(simple_pgrm.glid, modelCollider, "../assets/teapot.ply");


    scaleCollider = glm::scale(glm::mat4(1.0f), 0.38f*glm::vec3(1.5f, 1.0f, 1.0f));
    glm::mat4 rotCollider = glm::rotate(glm::mat4(1.0f), 3.14159266f/2.0f, glm::vec3(0.0f, 1.0f, 0.0f));
    transCollider = glm::translate(glm::mat4(1.0f), glm::vec3(-7.0f, 2.0f, 5.0f));
    modelCollider = rotCollider*transCollider*scaleCollider;
    MeshFromOBJ *anotherCollider2 = new MeshFromOBJ(simple_pgrm.glid, modelCollider, "../assets/heart.obj");


    sim->addCollider(ground);
    sim->addCollider(anotherCollider2);

    // Init GUI (imgui window)
    GUI *gui = new GUI(window);    

    // Rendering options
    glEnable(GL_DEPTH_TEST);

    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glClearColor(0.0, 0.0, 0.0, 1.0);

        gui->initNewFrame();

        float currentFrame = static_cast<float>(glfwGetTime());
        dt = currentFrame - lastFrame;
        lastFrame = currentFrame;

        keyboard_callback(window);

        // run simulation 
        sim->run(currentFrame);

        // camera matrices
        glm::mat4 projection = camera.projectionMatrix(w_width, w_height);
        glm::mat4 view = camera.viewMatrix();

        simple_pgrm.use();
        simple_pgrm.setMat4("projection", projection);
        simple_pgrm.setMat4("view", view);        
        simple_pgrm.setMat4("model", glm::mat4(1.0f));
        simple_pgrm.setVec3("camera_pos", camera.pos());
        GLenum wireframeMode = gui->colliderWireframe ? GL_LINE : GL_FILL;
        glPolygonMode(GL_FRONT_AND_BACK,  wireframeMode);
        anotherCollider2->draw();

        ground_pgrm.use();
        ground_pgrm.setMat4("projection", projection);
        ground_pgrm.setMat4("view", view);
        ground_pgrm.setMat4("model", glm::mat4(1.0f));
        glPolygonMode(GL_FRONT_AND_BACK,  GL_FILL);
        ground->draw(); 

        cloth_pgrm.use();
        cloth_pgrm.setMat4("projection", projection);
        cloth_pgrm.setMat4("view", view);
        cloth_pgrm.setMat4("model", glm::mat4(1.0f));
        simple_pgrm.setVec3("camera_pos", camera.pos());

        wireframeMode = gui->clothWireframe ? GL_LINE : GL_FILL;
        glPolygonMode(GL_FRONT_AND_BACK,  wireframeMode);
        cloth->draw();

        gui->buildWindow(sim, cloth); // TODO delete cast when implementing Implicit Solver <!>
        gui->render();

        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    glfwTerminate();
    return 0;

}

void windowResizeCallback(GLFWwindow *window, int width, int height)
{
    glViewport(0, 0, width, height);
} 

// -------------------------------------------------------
void mouseCallback(GLFWwindow* window, double xposIn, double yposIn)
{
    int status = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT);
    if (status == GLFW_PRESS)
    {
        float xpos = static_cast<float>(xposIn);
        float ypos = static_cast<float>(yposIn);

        if (firstMouse)
        {
            lastX = xpos;
            lastY = ypos;
            firstMouse = false;
            return;
        }

        float offsetX = xpos - lastX;
        float offsetY = lastY - ypos;

        lastX = xpos;
        lastY = ypos;

        camera.processRotation(offsetX, offsetY);
    }

    else if (status == GLFW_RELEASE) firstMouse = true;

}

void keyboard_callback(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::FORWARD, dt);
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::BACKWARD, dt);
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::LEFT, dt);
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::RIGHT, dt);
     if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::UP, dt);
}