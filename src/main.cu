#include "glad.h"
#include "../include/mesh.hcu"
#include "../include/camera.h"
#include "../include/simulation.h"
#include "../include/ui.h"
#include <iostream>

using namespace std;
float w_height = 1000.0;
float w_width = 1200.0;
float lastX;
float lastY;
bool firstMouse = true;

// callbacks
void windowResizeCallback(GLFWwindow *window, int width, int height);
void mouseCallback(GLFWwindow* window, double xpos, double ypos);
void keyboard_callback(GLFWwindow *window);


//Camera camera(glm::vec3(21.8819, 20.3187, 83.4559));
Camera camera(glm::vec3(100.0, 200., 700.0));

float dt = 0.0f;	// time between current frame and last frame
float lastFrame = 0.0f;
int main()
{
    // initializing OpenGL context using GLFW & GLAD
    if (!glfwInit())
    {
        std::cout << "Failed to init GLFW!" << std::endl;
        glfwTerminate();
        return -1;
    }
    GLFWwindow *window = glfwCreateWindow(static_cast<int>(w_width), static_cast<int>(w_height), "Cloth Simulation", 0, nullptr);
    if(!window)
    {
        std::cout << "Failed to create GLFW window!" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, windowResizeCallback);
    glfwSetCursorPosCallback(window, mouseCallback);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }


    std::cout << "MAIN" << std::endl << std::flush;


    // setting up CUDA context

    int device_count;
    cudaErrorCheck(hipGetDeviceCount(&device_count));
    if (device_count == 0) 
    {
        std::cout << "No CUDA devices found!" << std::endl << std::flush ;
        return -1;
    }


    // setting simulation objects (drawables)
    ShaderProgram ground_pgrm("../shaders/ground.vs", "../shaders/ground.fs");
    ShaderProgram sphere_pgrm("../shaders/sphere.vs", "../shaders/sphere.fs");
    ShaderProgram cloth_pgrm("../shaders/cloth.vs", "../shaders/cloth.fs");

    Plane *cloth = new Plane(cloth_pgrm.glid, 128);
    Plane *ground = new Plane(ground_pgrm.glid, 500, {false, false, false});
    ground->setPrimOpenGL(GL_LINES);
    Sphere *sphere = new Sphere(sphere_pgrm.glid);
    Simulation *sim = new Simulation(cloth);
    sim->addCollider(sphere);

    // Init GUI (imgui window)
    GUI *gui = new GUI(window); 

    

    // Rendering options
    glEnable(GL_DEPTH_TEST);
    //glEnable(GL_CULL_FACE);
    //glCullFace(GL_BACK);

    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glClearColor(0.0, 0.0, 0.0, 1.0);

        gui->initNewFrame();

        float currentFrame = static_cast<float>(glfwGetTime());
        dt = currentFrame - lastFrame;
        lastFrame = currentFrame;

        keyboard_callback(window);

        // run simulation 
        sim->run(currentFrame);

        // camera matrices
        glm::mat4 projection = camera.projectionMatrix(w_width, w_height);
        glm::mat4 view = camera.viewMatrix();

        sphere_pgrm.use();
        sphere_pgrm.setMat4("projection", projection);
        sphere_pgrm.setMat4("view", view);
        glm::mat4 model = glm::scale(glm::mat4(1.0f), glm::vec3(5.0f));        
        sphere_pgrm.setMat4("model", model);
        glPolygonMode(GL_FRONT_AND_BACK,  GL_LINE);
        sphere->draw();

        ground_pgrm.use();
        ground_pgrm.setMat4("projection", projection);
        ground_pgrm.setMat4("view", view);
        model = glm::scale(glm::mat4(1.0f), 25.0f*glm::vec3(1.0f, 0.0f, 1.0f));
        model = glm::translate(model, -250.0f*glm::vec3(1.0f, 0.0f, 1.0f));
        ground_pgrm.setMat4("model", model);
        glPolygonMode(GL_FRONT_AND_BACK,  GL_LINE);
        ground->draw(); 

        cloth_pgrm.use();
        cloth_pgrm.setMat4("projection", projection);
        cloth_pgrm.setMat4("view", view);
        model = glm::scale(glm::mat4(1.0), glm::vec3(2.0f));
        model = glm::translate(model, glm::vec3(0.0f, 100.0f, 0.0f));
        cloth_pgrm.setMat4("model", model);

        glPolygonMode(GL_FRONT_AND_BACK,  GL_FILL);
        cloth->draw();

        gui->buildWindow(sim, cloth); // TODO delete cast when implementing Implicit Solver <!>
        gui->render();

        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    glfwTerminate();
    return 0;

}

void windowResizeCallback(GLFWwindow *window, int width, int height)
{
    glViewport(0, 0, width, height);
} 

// -------------------------------------------------------
void mouseCallback(GLFWwindow* window, double xposIn, double yposIn)
{
    int status = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT);
    if (status == GLFW_PRESS)
    {
        float xpos = static_cast<float>(xposIn);
        float ypos = static_cast<float>(yposIn);

        if (firstMouse)
        {
            lastX = xpos;
            lastY = ypos;
            firstMouse = false;
            return;
        }

        float offsetX = xpos - lastX;
        float offsetY = lastY - ypos;

        lastX = xpos;
        lastY = ypos;

        camera.processRotation(offsetX, offsetY);
    }

    else if (status == GLFW_RELEASE) firstMouse = true;

}

void keyboard_callback(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::FORWARD, dt);
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::BACKWARD, dt);
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::LEFT, dt);
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::RIGHT, dt);
     if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
        camera.processTranslation(Camera_Movement::UP, dt);
}